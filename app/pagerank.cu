#include "hip/hip_runtime.h"
#include <stdio.h>

#include "LightTraffic.h"
#include "walk/NodeSampler.h"

const float alpha = 0.85;

typedef struct walker {
    nodeId current;
    nodeId step;
} Walker;

struct App { 
public:
    nodeId *_value;

    nodeId _numNode;
    walkId _numWalker;
    nodeId _maxStep;

    __device__ bool terminated(Walker &walker) {
        assert(walker.step <= _maxStep);
        return walker.step == 0;
    }

    __device__ void update(Walker &walker, const edgeId *rowptr, const nodeId *col, const nodeId nodeOffset, const edgeId edgeOffset, GPURandState &state) {
        atomicAdd(&_value[walker.current], 1);
        
        walker.step -= 1;
        
        if (terminated(walker)) {
            return;
        }

        if (hiprand_uniform_double(&state) > alpha) {
            walker.current = (nodeId)uniform_discrete_distribution(state, _numNode);
        }
        else {
            edgeId firstEdge = rowptr[walker.current - nodeOffset];
            edgeId deg = rowptr[walker.current + 1 - nodeOffset] - firstEdge;

            walker.current = col[firstEdge + (edgeId)uniform_discrete_distribution(state, deg) - edgeOffset];
        }
    }
};

struct AppManager {
private:
    CPUBuffer<nodeId> _h_value;
    GPUBuffer<nodeId> _d_value;
    GPUBuffer<App> _app;

    nodeId _numNode;

    int _runs;
    int _currentRun{0};

public:
    walkId _numWalker;
    nodeId _maxStep;
    GPUNodeSampler<Walker> _gpuInit;

    AppManager(nodeId numNode, Config &config, int gpuId): _h_value(numNode), _d_value(numNode, gpuId), _app(1, gpuId),
       _numNode(numNode), _runs(config._runs), _numWalker(config._numWalker), _maxStep(config._step), _gpuInit(_numWalker, numNode, _maxStep)
    {
        App app{_d_value.ptr(), _numNode, _numWalker, _maxStep};
        CPUBuffer<App> host_app(1, &app);
        host_app.to(_app);
    }

    void reduce() {
        if (_currentRun >= _runs)
            _d_value.to(_h_value);
    }

    int epochs() {
        return _runs;
    }

    void createWalkers(GPUWalkManager<App, Walker> &gpuWalkman, WalkManager<Walker> &cpuWalkman, CUDAStream &stream) {
        _currentRun++;
        if (_currentRun > _runs) {
            return;
        }

        gpuWalkman.insert(_gpuInit, cpuWalkman, stream);
    }

    bool check() {
        size_t sum = 0;
        for (nodeId i = 0; i < _numNode; i++) {
            sum += _h_value[i];
        }
        size_t expected = (size_t)(_numWalker) * (size_t)(_maxStep) * (size_t)(_runs);
        printf("sum: %zu, %zu\n", sum, expected);
        return sum == expected;
    }

    float result(nodeId i) {
        return (_h_value[i] / (_numWalker * _runs / (_numNode + 0.0))) / (_maxStep + 0.0);
    }

    auto GPUApp() {
        return _app.ptr();
    }
};

int main(int argc, char* argv[])
{   
    Config config(argc, argv);
    PartitionStrategy &partitions = *(config._partitions);

    int gpuId = 0;
    AppManager rw(partitions.numNode(), config, gpuId);

    LightTraffic<AppManager, App, Walker> program(config, rw);
    program.start(rw);

    edgeId *rowptr = program._cpu->_graph.rowptr().ptr();
    for (nodeId i = 0, printCount = 0; i < partitions.numNode() && printCount < 20; i++) {
        edgeId deg = rowptr[i + 1] - rowptr[i];
        if (deg > 1000) {
            printf("node %u, degree %lu, value %f\n", i, deg, rw.result(i));
            printCount++;
        }
    }

    printf("node 0, degree %lu, value %f\n", rowptr[1], rw.result(0));

    bool check = rw.check();
    printf(check? "result correct\n": "result incorrect\n");

    if (!check) {
        printf("[Warning] ill-conditioned graph such as Yahoo has a very high-degree vertex whose value could exceed the max value of uint32.\n");
    }

    return 0;
}
