#include "hip/hip_runtime.h"
#include <stdio.h>

#include "LightTraffic.h"
#include "walk/SingleSource.h"

typedef struct walker {
    nodeId current;
    nodeId step;
} Walker;

struct App { 
public:
    nodeId *_value;

    nodeId _numNode;
    walkId _numWalker;

    __device__ bool terminated(Walker &walker) {
        return walker.step == 0;
    }

    __device__ void update(Walker &walker, const edgeId *rowptr, const nodeId *col, const nodeId nodeOffset, const edgeId edgeOffset, GPURandState &state) {
        if (walker.step == 1) {
            atomicAdd(&_value[walker.current], 1);
        }

        walker.step -= 1;
        
        if (terminated(walker)) {
            return;
        }
        
        edgeId firstEdge = rowptr[walker.current - nodeOffset];
        edgeId deg = rowptr[walker.current + 1 - nodeOffset] - firstEdge;

        if (deg > 0)
            walker.current = col[firstEdge + (edgeId)uniform_discrete_distribution(state, deg) - edgeOffset];
        else {
            atomicAdd(&_value[walker.current], 1);
            walker.step = 0;
        }
    }
};

struct AppManager {
private:
    CPUBuffer<nodeId> _h_value;
    GPUBuffer<nodeId> _d_value;
    GPUBuffer<App> _app;

    nodeId _numNode;

public:
    walkId _numWalker;
    GPUSpecifiedNode<Walker> _gpuInit;

    AppManager(nodeId numNode, Config &config, int gpuId): _h_value(numNode), _d_value(numNode, gpuId), _app(1, gpuId),
       _numNode(numNode), _numWalker(config._numWalker), _gpuInit(_numWalker, config._source, config._prob)
    {
        App app{_d_value.ptr(), _numNode, _numWalker};
        CPUBuffer<App> host_app(1, &app);
        host_app.to(_app);
    }

    void reduce() {
        _d_value.to(_h_value);
    }

    int epochs() {
        return 1;
    }

    void createWalkers(GPUWalkManager<App, Walker> &gpuWalkman, WalkManager<Walker> &cpuWalkman, CUDAStream &stream) {
        gpuWalkman.insert(_gpuInit, cpuWalkman, stream);
    }

    bool check() {
        size_t sum = 0;
        for (nodeId i = 0; i < _numNode; i++) {
            sum += _h_value[i];
        }
        size_t expected = _numWalker;
        printf("sum: %lu, %lu\n", sum, expected);
        return sum == expected;
    }

    float result(nodeId i) {
        return (_h_value[i] / (_numWalker / (_numNode + 0.0)));
    }

    auto GPUApp() {
        return _app.ptr();
    }
};

int main(int argc, char* argv[])
{   
    Config config(argc, argv);
    PartitionStrategy &partitions = *(config._partitions);

    int gpuId = 0;
    AppManager rw(partitions.numNode(), config, gpuId);

    LightTraffic<AppManager, App, Walker> program(config, rw);
    program.start(rw);

    nodeId target = 2;
    printf("PageRank value of node %u: %f\n", target, rw.result(target));

    printf(rw.check()? "result correct\n": "result incorrect\n");

    return 0;
}
